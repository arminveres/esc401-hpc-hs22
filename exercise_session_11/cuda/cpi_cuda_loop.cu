// Using CUDA device to calculate pi
#include <hip/hip_runtime.h>

#include <array>
#include <cstdint>
#include <cstdio>

extern "C" double getTime(void);

#define NBIN 1000000000  // Number of bins
// #define NUM_BLOCK (2 * 56)  // Number of thread blocks
// #define NUM_THREAD (2 * 8)  // Number of threads per block

// Kernel that executes on the CUDA device
__global__ void cal_pi(double *sum, int nbin, double step, int nthreads, int nblocks) {
    int i;
    double x;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;  // Sequential thread index across the blocks
    for (i = idx; i < nbin; i += nthreads * nblocks) {
        x = (i + 0.5) * step;
        sum[idx] += 4.0 / (1.0 + x * x);
    }
}

// Main routine that executes on the host
int main(void) {
    static constexpr std::array<uint32_t, 8> blocks = {60, 120, 180, 240, 300, 360, 420, 600};
    static constexpr std::array<uint32_t, 10> threads = {16, 32,  48,  64,  80,
                                                         96, 112, 128, 144, 160};
    auto counter = 0;
    for (auto i = 0; i < blocks.size(); ++i) {
        for (auto j = 0; j < threads.size(); ++j) {
            const auto NUM_BLOCK = blocks[i];
            const auto NUM_THREAD = threads[j];

            dim3 dimGrid(NUM_BLOCK, 1, 1);    // Grid dimensions
            dim3 dimBlock(NUM_THREAD, 1, 1);  // Block dimensions
            double *sumHost, *sumDev;         // Pointer to host & device arrays
            double pi = 0;
            int tid;

            double step = 1.0 / NBIN;                               // Step size
            size_t size = NUM_BLOCK * NUM_THREAD * sizeof(double);  // Array memory size
            sumHost = (double *)malloc(size);                       //  Allocate array on host
            hipMalloc((void **)&sumDev, size);                     // Allocate array on device
            double start = getTime();

            // Initialize array in device to 0
            hipMemset(sumDev, 0, size);
            // Do calculation on device
            cal_pi<<<dimGrid, dimBlock>>>(sumDev, NBIN, step, NUM_THREAD,
                                          NUM_BLOCK);  // call CUDA kernel
            // Retrieve result from device and store it in host array
            hipMemcpy(sumHost, sumDev, size, hipMemcpyDeviceToHost);
            for (tid = 0; tid < NUM_THREAD * NUM_BLOCK; tid++) pi += sumHost[tid];
            pi *= step;

            // Print results
            double delta = getTime() - start;
            printf("BLOCK: %d, THREAD: %d, PI = %.16g computed in %.4g seconds\n", NUM_BLOCK,
                   NUM_THREAD, pi, delta);
            // Cleanup
            free(sumHost);
            hipFree(sumDev);
            counter++;
        }
    }
    printf("%d\n", counter);
    return 0;
}